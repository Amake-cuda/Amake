#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : TESTKERNELCUMP.cu
 Author      : amakje
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <gmp.h>
#include <cump/cump.cuh>



//static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
//#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

using cump::mpf_array_t;


__global__ void cump_scal_kernel(int n, mpf_array_t alpha, mpf_array_t x) {
    using namespace cump;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < n) {
        mpf_mul(x[idx], alpha[0], x[idx]);
        idx += gridDim.x * blockDim.x;
    }
}






int main ()
{
	int n=256;
	int prec = 20;
	int seed =20;
	 mpf_set_default_prec(prec);
	    cumpf_set_default_prec(prec);

	    //Execution configuration
	    int threads = 64;
	    int blocks = n / threads + (n % threads ? 1 : 0);
	    gmp_randstate_t  rstate;

	    //Host data
	    mpf_t *hx = new mpf_t[n];
	    mpf_t halpha;

	    //GPU data
	    cumpf_array_t dx;
	    cumpf_array_t dalpha;

	    cumpf_array_init(dx, n);
	    cumpf_array_init(dalpha, 1);


				  gmp_randinit_default (rstate);
	      		  gmp_randseed_ui (rstate, seed);

	      		mpf_init (halpha);
			  mpf_urandomb (halpha, rstate, prec);

	      		  for (int i = 0;  i < n;  ++i)
	      		    {
	      		      mpf_init (hx [i]);
	      		      mpf_urandomb (hx [i], rstate, prec);

	      		    }
	      		gmp_randclear (rstate);
	      		gmp_printf ("HX: %.70Ff \n", hx[0]);
	      		gmp_printf ("HALPHA: %.70Ff \n", halpha);

	    //Copying alpha to the GPU
	    cumpf_array_set_mpf(dalpha, &halpha, 1);

	        cumpf_array_set_mpf(dx, hx, n);


	        cump_scal_kernel<<<blocks, threads>>>(n, dalpha, dx);
	        hipError_t errSync  = hipGetLastError();
	        	hipError_t errAsync = hipDeviceSynchronize();
	        	if (errSync != hipSuccess)
	        	  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	        	if (errAsync != hipSuccess)
	        	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));


	    //Copying to the host
	    mpf_array_set_cumpf(hx, dx, n);
	    for(int i = 1; i < n; i ++){
	        mpf_add(hx[0], hx[i], hx[0]);
	    }
	    gmp_printf ("HX RESULT: %.70Ff \n", hx[0]);

	    //Cleanup
	    mpf_clear(halpha);
	    for(int i = 0; i < n; i ++){
	        mpf_clear(hx[i]);
	    }
	    delete [] hx;
	    cumpf_array_clear(dalpha);
	    cumpf_array_clear(dx);


	return 0;
}


